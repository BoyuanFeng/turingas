#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main() {
  int * input;
  int * output;
  int * input_d;
  int * output_d;

  input  = (int*)malloc(2 * sizeof(int));
  output = (int*)malloc(2 * sizeof(int));

  input[0] = 0;
  output[0] = 0;

  hipMalloc((void**)&input_d,  2 * sizeof(int));
  hipMalloc((void**)&output_d, 2 * sizeof(int));

  hipMemcpy(input_d, input, 2 * sizeof(int), hipMemcpyHostToDevice);
  
  hipModule_t module;
  hipFunction_t kernel;

  hipModuleLoad(&module, "a.cubin");
  hipModuleGetFunction(&kernel, module, "kern");

  void * args[2] = {&input_d, &output_d};
  hipModuleLaunchKernel(kernel, 1, 1, 1, 
                 32, 1, 1, 
                 0, 0, args, 0);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, 2 * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(input, input_d, 2 * sizeof(int), hipMemcpyDeviceToHost);

  cout << "Result:\t" << (uint)output[0] << endl;


  return 0;
  
}
