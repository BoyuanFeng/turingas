#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main() {
  int * input;
  int * output;
  int * input_d;
  int * output_d;

  input  = (int*)malloc(sizeof(int));
  output = (int*)malloc(sizeof(int));

  input[0] = 10;
  output[0] = 0;

	cout << "Before the copy kernel." << endl;
	cout << "Input value:\t" << input[0] << endl;
  cout << "Output value:\t" << (uint)output[0] << endl;

  hipMalloc((void**)&input_d,  sizeof(int));
  hipMalloc((void**)&output_d, sizeof(int));

  hipMemcpy(input_d, input, sizeof(int), hipMemcpyHostToDevice);
  
  hipModule_t module;
  hipFunction_t kernel;

  hipModuleLoad(&module, "copy.cubin");
  hipModuleGetFunction(&kernel, module, "kern");

  void * args[2] = {&input_d, &output_d};
  hipModuleLaunchKernel(kernel, 1, 1, 1, 
                 1, 1, 1, 
                 0, 0, args, 0);
  hipDeviceSynchronize();

  hipMemcpy(output, output_d, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(input, input_d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "After the copy kernel." << endl;
	cout << "Input value:\t" << input[0] << endl;
  cout << "Output value:\t" << output[0] << endl;


  return 0;
  
}
